#include "hip/hip_runtime.h"
/* Copyright 2021 The LightSeq Team
   Copyright NVIDIA/apex
   This apex_adam_cuda_kernel is adapted from NVIDIA/apex
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cmath>

#include "ATen/ATen.h"
#include "ATen/AccumulateType.h"
#include "ATen/TensorUtils.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/cuda/detail/IndexUtils.cuh"
#include "ATen/cuda/Exceptions.h"
#include "../include/fused_adan_kernel.h"
#include "../include/multi_tensor_apply.cuh"

// void adan(at::Tensor& p, at::Tensor& p_copy, at::Tensor& g, at::Tensor& exp_avg, 
//           at::Tensor& exp_avg_sq, at::Tensor& exp_avg_diff,
//           at::Tensor& pre_g, float beta1, float beta2, float beta3, 
//           float bias_correction1, float bias_correction2, float bias_correction3_sqrt, 
//           float lr, float decay, float eps, bool no_prox, float grad_scale);

template <typename T, typename GRAD_T>
__global__ void adan_cuda_kernel(
    T* __restrict__ p,
    GRAD_T* __restrict__ p_copy,  // For mixed precision training, pass NULL if
                                  // not needed
    const GRAD_T* __restrict__ g, T* __restrict__ exp_avg, T* __restrict__ exp_avg_sq, T* __restrict__ exp_avg_diff,
    const GRAD_T* __restrict__ pre_g, const float b1, const float b2, const float b3, 
    const float bias_correction1, const float bias_correction2, const float bias_correction3_sqrt,
    const float lr, const float decay, const float eps, const bool no_prox, const float grad_scale
    ){
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_id >= total_size) return;

    T scaled_grad = g[global_id] / grad_scale;

    diff = scaled_grad - pre_g[global_id];
    update = scaled_grad + b2 * diff;

    // exp_avg.mul_(beta1).add_(grad, alpha=1 - beta1)  # m_t
    exp_avg[global_id] = b1 * exp_avg[global_id] + (1 - b1) * scaled_grad;

    // exp_avg_diff.mul_(beta2).add_(diff, alpha=1 - beta2)  # diff_t
    exp_avg_diff[global_id] = b2 * exp_avg_diff[global_id] + (1 - b2) * diff;

    // exp_avg_sq.mul_(beta3).addcmul_(update, update, value=1 - beta3)  # n_t
    exp_avg_sq[global_id] = b3 * exp_avg_sq[global_id] + (1 - b3) * update * update;

    // denom = ((exp_avg_sq).sqrt() / bias_correction3_sqrt).add_(eps)
    // update = ((exp_avg / bias_correction1 + beta2 * exp_avg_diff / bias_correction2)).div_(denom)
    float denom;
    denom = sqrtf(exp_avg_sq[global_id]) / bias_correction3_sqrt + eps;
    update = (exp_avg[global_id] / bias_correction1 + b2 * exp_avg_diff[global_id] / bias_correction2) / denom;
    
    if (no_prox){
        // param.mul_(1 - lr * weight_decay)
        // param.add_(update, alpha=-lr)
        p[global_id] = p[global_id] * (1 - lr * decay) + update * (-lr);
    }else{
        // param.add_(update, alpha=-lr)
        // param.div_(1 + lr * weight_decay)
        p[global_id] = p[global_id] + update * (-lr) / (1 + lr * decay);
    } 
    if (p_copy != NULL) p_copy[global_id] = (GRAD_T)p[global_id];
}

template <>
__global__ void adan_cuda_kernel<float, float>(
    float* __restrict__ p,
    float* __restrict__ p_copy,  // For mixed precision training, pass NULL if
                                  // not needed
    const float* __restrict__ g, float* __restrict__ exp_avg, float* __restrict__ exp_avg_sq, float* __restrict__ exp_avg_diff,
    const float* __restrict__ pre_g, const float b1, const float b2, const float b3, 
    const float bias_correction1, const float bias_correction2, const float bias_correction3_sqrt,
    const float lr, const float decay, const float eps, const bool no_prox, const float grad_scale){

        int global_id = blockIdx.x * blockDim.x + threadIdx.x;

        if (global_id * 4 >= total_size) return;

        const float4* g4_ptr = reinterpret_cast<const float4*>(g);
        const float4* pre_g4_ptr = reinterpret_cast<const float4*>(pre_g);
        float4* exp_avg4_ptr = reinterpret_cast<float4*>(exp_avg);
        float4* exp_avg_sq4_ptr = reinterpret_cast<float4*>(exp_avg_sq);
        float4* exp_avg_diff4_ptr = reinterpret_cast<float4*>(exp_avg_diff);

        const float4 g4 = g4_ptr[global_id];
        const float4 pre_g4 = pre_g4_ptr[global_id];
        float4 exp_avg4 = exp_avg4_ptr[global_id];
        float4 exp_avg_sq4 = exp_avg_sq4_ptr[global_id];
        float4 exp_avg_diff4 = exp_avg_diff4_ptr[global_id];

        float4 new_exp_avg4;
        float4 new_exp_avg_sq4;
        float4 new_exp_avg_diff4;

        float scaled_grad1 = g4.x / grad_scale;
        float scaled_grad2 = g4.y / grad_scale;
        float scaled_grad3 = g4.z / grad_scale;
        float scaled_grad4 = g4.w / grad_scale;

        float diff1 = scaled_grad1 - pre_g4.x;
        float diff2 = scaled_grad2 - pre_g4.y;
        float diff3 = scaled_grad3 - pre_g4.z;
        float diff4 = scaled_grad4 - pre_g4.w;

        float update1 = scaled_grad1 + b2 * diff1;
        float update2 = scaled_grad2 + b2 * diff2;
        float update3 = scaled_grad3 + b2 * diff3;
        float update4 = scaled_grad4 + b2 * diff4;

        new_exp_avg4.x = b1 * exp_avg4.x + (1 - b1) * scaled_grad1;
        new_exp_avg4.y = b1 * exp_avg4.y + (1 - b1) * scaled_grad2;
        new_exp_avg4.z = b1 * exp_avg4.z + (1 - b1) * scaled_grad3;
        new_exp_avg4.w = b1 * exp_avg4.w + (1 - b1) * scaled_grad4;

        new_exp_avg_sq4.x = b3 * exp_avg_sq4.x + (1 - b3) * update1 * update1;
        new_exp_avg_sq4.y = b3 * exp_avg_sq4.y + (1 - b3) * update2 * update2;
        new_exp_avg_sq4.z = b3 * exp_avg_sq4.z + (1 - b3) * update3 * update3;
        new_exp_avg_sq4.w = b3 * exp_avg_sq4.w + (1 - b3) * update4 * update4;

        new_exp_avg_diff4.x = b2 * exp_avg_diff4.x + (1 - b2) * diff1;
        new_exp_avg_diff4.y = b2 * exp_avg_diff4.y + (1 - b2) * diff2;
        new_exp_avg_diff4.z = b2 * exp_avg_diff4.z + (1 - b2) * diff3;
        new_exp_avg_diff4.w = b2 * exp_avg_diff4.w + (1 - b2) * diff4;

        float4 denom4;
        denom4.x = sqrt(new_exp_avg_sq4.x - new_exp_avg_diff4.x * new_exp_avg_diff4.x / b2) + eps;
        denom4.y = sqrt(new_exp_avg_sq4.y - new_exp_avg_diff4.y * new_exp_avg_diff4.y / b2) + eps;
        denom4.z = sqrt(new_exp_avg_sq4.z - new_exp_avg_diff4.z * new_exp_avg_diff4.z / b2) + eps;
        denom4.w = sqrt(new_exp_avg_sq4.w - new_exp_avg_diff4.w * new_exp_avg_diff4.w / b2) + eps;

        // update = (exp_avg[global_id] / bias_correction1 + b2 * exp_avg_diff[global_id] / bias_correction2) / denom;
        update1 = (new_exp_avg4.x / bias_correction1 + b2 * new_exp_avg_diff4.x / bias_correction2) / denom4.x;
        update2 = (new_exp_avg4.y / bias_correction1 + b2 * new_exp_avg_diff4.y / bias_correction2) / denom4.y;
        update3 = (new_exp_avg4.z / bias_correction1 + b2 * new_exp_avg_diff4.z / bias_correction2) / denom4.z;
        update4 = (new_exp_avg4.w / bias_correction1 + b2 * new_exp_avg_diff4.w / bias_correction2) / denom4.w;

        if (no_prox){
            // p[global_id] = p[global_id] * (1 - lr * decay) + update * (-lr);
            new_p4.x = p4.x * (1 - lr * decay) + update1 * (-lr);
            new_p4.y = p4.y * (1 - lr * decay) + update2 * (-lr);
            new_p4.z = p4.z * (1 - lr * decay) + update3 * (-lr);
            new_p4.w = p4.w * (1 - lr * decay) + update4 * (-lr);
        }else{
            // p[global_id] = p[global_id] + update * (-lr) / (1 + lr * decay);
            new_p4.x = p4.x + update1 * (-lr) / (1 + lr * decay);
            new_p4.y = p4.y + update2 * (-lr) / (1 + lr * decay);
            new_p4.z = p4.z + update3 * (-lr) / (1 + lr * decay);
            new_p4.w = p4.w + update4 * (-lr) / (1 + lr * decay);
        }   

        p4_ptr[global_id] = new_p4;
        exp_avg4_ptr[global_id] = new_exp_avg4;
        exp_avg_sq4_ptr[global_id] = new_exp_avg_sq4;
        exp_avg_diff4_ptr[global_id] = new_exp_avg_diff4;
}

void fused_adan_cuda(at::Tensor& p, at::Tensor& p_copy, at::Tensor& g, at::Tensor& exp_avg, 
          at::Tensor& exp_avg_sq, at::Tensor& exp_avg_diff,
          at::Tensor& pre_g, float beta1, float beta2, float beta3, 
          float bias_correction1, float bias_correction2, float bias_correction3_sqrt, 
          float lr, float decay, float eps, bool no_prox, float grad_scale){
    // Get tensor size
    int total_size = p.numel();
    AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p),
              "parameter tensor is too large to be indexed with int32");
    
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    if (g.scalar_type() == at::ScalarType::Half) {
        const int block_dim = 1024;
        int grid_dim = ((total_size + block_dim - 1) / block_dim);
        const dim3 blocks(grid_dim);
        // all other values should be fp32 for half gradients
        AT_ASSERTM(p.scalar_type() == at::ScalarType::Float,
                  "expected parameter to be of float type");
        // dispatch is done on the gradient type
        using namespace at;  // prevents "toString is undefined" errors
        DISPATCH_FLOAT_AND_HALF(
            g.scalar_type(), 0, "adan_cuda_kernel",
            using accscalar_t = at::acc_type<scalar_t_0, true>;
            adan_cuda_kernel<accscalar_t, scalar_t_0>
            <<<blocks, block_dim, 0, stream>>>(
                p.DATA_PTR<accscalar_t>(),
                p_copy.numel() ? p_copy.DATA_PTR<scalar_t_0>() : NULL,
                g.DATA_PTR<scalar_t_0>(), exp_avg.DATA_PTR<accscalar_t>(), exp_avg_sq.DATA_PTR<accscalar_t>(),exp_avg_diff.DATA_PTR<accscalar_t>(), 
                pre_g.DATA_PTR<scalar_t_0>(), 
                beta1, beta2, beta3, bias_correction1, bias_correction2, bias_correction3_sqrt, 
                lr, decay, eps, no_prox, grad_scale
                );
            );
    } else {
        using namespace at;
        const int block_dim = 1024;
        int grid_dim = ((total_size + block_dim - 1) / block_dim) >> 2;
        if (grid_dim == 0) grid_dim = 1;
        const dim3 blocks(grid_dim);
        DISPATCH_DOUBLE_AND_FLOAT(
            g.scalar_type(), 0, "adan_cuda_kernel",
            adan_cuda_kernel<scalar_t_0, scalar_t_0>
            <<<blocks, block_dim, 0, stream>>>(
                p.DATA_PTR<scalar_t_0>(),
                NULL,
                g.DATA_PTR<scalar_t_0>(), exp_avg.DATA_PTR<scalar_t_0>(), exp_avg_sq.DATA_PTR<scalar_t_0>(),exp_avg_diff.DATA_PTR<scalar_t_0>(), 
                pre_g.DATA_PTR<scalar_t_0>(), 
                beta1, beta2, beta3, bias_correction1, bias_correction2, bias_correction3_sqrt, 
                lr, decay, eps, no_prox, grad_scale
            );
        );
    }
    AT_CUDA_CHECK(hipGetLastError());
}

